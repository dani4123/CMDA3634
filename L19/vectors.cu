#include "hip/hip_runtime.h"
#include
#include
#include
#include

#include "hip/hip_runtime.h"

int main(argc, char **argv){

	//get vector size from command line argument
	int N = atoi(argv[1]);

	//seed RNG
	double seed = clock();
	srand48(seed);

	double *h_a, *h_b, *h_c; //host vectors

	//allocate storage
	h_a = (double *) molloc(N*sizeof(double));	
	h_b = (double *) molloc(N*sizeof(double));
	h_c = (double *) molloc(N*sizeof(double));

	//populate a and b
	for(int n = 0; n<N; n++){
	h_a[n] = drand48();
	h_b[n] = drand48();
	}


	double hostStart = clock();

	//c = a+b
	for(int n =0; n<N; n++){
		h_c[n] = h_a[n] + h_b[n];
	}

	double hostEnd = clock();
	double hostTime = (hostEnd - hostStart)/(double)CLOCKS_PER_SEC;
	
	printf("the host took %f seconds to add a and b \n", hostTime);

	//device arrays
	double *d_a, *d_b, *d_c;

	hipMalloc(&d_a,N*sizeof(double));	
	hipMalloc(&d_b,N*sizeof(double));
	hipMalloc(&d_c,N*sizeof(double));

	//copy data from the host to the device
	hipMemcpy(d_a, h_a,N*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b,N*sizeof(double),hipMemcpyHostToDevice);
	//hipMemcpy(d_a, h_a,N*sizeof(double),hipMemcpyHostToDevice);
	

	//at this point the data is allocated and populated on the device

	int Nthreads = atoi(argv[2]); //get the number of threads per block from command line
	

	kernelAddVectors <<< ,Nthreads >>>(d_a, d_b, d_c);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);


}
}
